#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Decoder.h"
#include <vector>

hipError_t count_gt_leq_with_cuda(std::vector<long long int> data, long long int target, int* leq, int* gt);

__global__ void count_gt_leq_kernel(long long int* data, long long int target, int* res_leq, int* res_gt, unsigned int size)
{
    int elem_index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (elem_index >= size)
        return;

    long long int value = data[elem_index];
    printf("%d | [%d->%d]=%lld\n", threadIdx.x, elem_index, size, value);
    if (abs(value) > target) res_gt[threadIdx.x]++;
    else res_leq[threadIdx.x]++;
}

int main()
{
    auto data = Decode("file2.wav");
    printf("File length is %d", data.size());
    long long int target = 16000;
    int leq = 0; 
    int gt = 0;

    hipError_t cudaStatus = count_gt_leq_with_cuda(data, target, &leq, &gt);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "count_gt_leq_with_cuda failed! ");
        return 1;
    }

    printf("leq = %d | gt = %d", leq, gt);
    getchar();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t count_gt_leq_with_cuda(std::vector<long long int> data, long long int target, int* leq, int* gt)
{
    long long int* dev_data = 0;
    int *dev_leq = 0, *dev_gt = 0;
    auto size = data.size();
    int *host_leq = new int[size], *host_gt = new int[size];
    const int THREAD_COUNT = 1024;
    dim3 gridSize = dim3(size / THREAD_COUNT + 1);
    dim3 blockSize = dim3(THREAD_COUNT);

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Выделяем память на GPU для входных данных
    cudaStatus = hipMalloc((void**)&dev_data, size * sizeof(long long int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Выделяем память на GPU для выходных данных
    cudaStatus = hipMalloc((void**)&dev_leq, THREAD_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_gt, THREAD_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    // Копируем входные данные в буффер GPU
    cudaStatus = hipMemcpy(dev_data, data.data(), size * sizeof(long long int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    count_gt_leq_kernel<<<gridSize, blockSize>>>(dev_data, target, dev_leq, dev_gt, size);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "count_gt_leq_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Ждем выполнения всех задач на GPU
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Копируем значения из буффера gpu на хост
    cudaStatus = hipMemcpy(host_leq, dev_leq, THREAD_COUNT * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(host_gt, dev_gt, THREAD_COUNT * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Собираем результат с ядер
    for (int i = 0; i < THREAD_COUNT; i++)
    {
        (*gt) += host_gt[i];
        (*leq) += host_leq[i];
    }

Error:
    hipFree(dev_data);
    hipFree(dev_leq);
    hipFree(dev_gt);
    free(host_gt);
    free(host_leq);
    return cudaStatus;
}
